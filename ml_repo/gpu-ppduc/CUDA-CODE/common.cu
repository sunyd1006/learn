#ifndef COMMON_CU_H
#define COMMON_CU_H

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

// // Read a 24-bit/pixel BMP file into a 1D linear array.
// // Allocate memory to store the 1D image and return its pointer.
// uch *ReadBMPlin(char* fn)
// {
// 	static uch *Img;
// 	FILE* f = fopen(fn, "rb");
// 	if (f == NULL){	printf("\n\n%s NOT FOUND\n\n", fn);	exit(EXIT_FAILURE); }

// 	uch HeaderInfo[54];
// 	fread(HeaderInfo, sizeof(uch), 54, f); // read the 54-byte header
// 	// extract image height and width from header
// 	int width = *(int*)&HeaderInfo[18];			ip.Hpixels = width;
// 	int height = *(int*)&HeaderInfo[22];		ip.Vpixels = height;
// 	int RowBytes = (width * 3 + 3) & (~3);		ip.Hbytes = RowBytes;
// 	//save header for re-use
// 	memcpy(ip.HeaderInfo, HeaderInfo,54);
// 	printf("\n Input File name: %17s  (%u x %u)   File Size=%u", fn,
// 			ip.Hpixels, ip.Vpixels, IMAGESIZE);
// 	// allocate memory to store the main image (1 Dimensional array)
// 	Img  = (uch *)malloc(IMAGESIZE);
// 	if (Img == NULL) return Img;      // Cannot allocate memory
// 	// read the image from disk
// 	fread(Img, sizeof(uch), IMAGESIZE, f);
// 	fclose(f);
// 	return Img;
// }


#endif COMMON_CU_H

